#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>


#define RECONSTRUCTION_FUNCTION(name, code) \
__global__ void name(\
    torch::PackedTensorAccessor<float,3,torch::RestrictPtrTraits,size_t> img,\
    const torch::PackedTensorAccessor<float,3,torch::RestrictPtrTraits,size_t>  grad_x){\
    int index = blockIdx.x * blockDim.x + threadIdx.x;\
    int stride = blockDim.x * gridDim.x;\
    int col = blockIdx.y;\
    code}

RECONSTRUCTION_FUNCTION(lr_kernel,{
    for(int y = index+1; y < img.size(1)-1; y+=stride){
        for(int x = 1; x < img.size(2)-1; x++){
            img[col][y][x] = (img[col][y][x] + img[col][y][x - 1] + grad_x[col][y][x - 1]) / 2;
        }
    }
})

RECONSTRUCTION_FUNCTION(rl_kernel,{
    for(int y = index+1; y < img.size(1)-1; y+=stride){
        for(int x = img.size(2)-2; x > 0; x--){
            img[col][y][x] = (img[col][y][x] + img[col][y][x + 1] - grad_x[col][y][x]) / 2;
        }
    }
})


void step_cuda(int step, torch::Tensor img, torch::Tensor grad){
    //const auto img_width = img.size(1)
    // TODO dim3 to parallelize colors
    dim3 num_blocks(4,3);

    switch(step%4){
    case 0:
        lr_kernel<<<num_blocks,1024>>>(
            img.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>(),
            grad.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>());
        break;
    case 1:
        //tb_kernel<<<num_blocks,1024>>>(
        //    img.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>(),
        //    grad_x.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>());
        break;
    case 2:
        rl_kernel<<<num_blocks,1024>>>(
            img.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>(),
            grad.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>());
        break;
    default:
        //bt_kernel<<<num_blocks,1024>>>(
        //    img.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>(),
        //    grad_x.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>());
        break;
    }
}
