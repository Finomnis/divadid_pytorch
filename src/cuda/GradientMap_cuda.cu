#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


// A macro to remove all the messy boilerplate code.
// Defines the function name and sets up the convenience variables 'index', 'stride' and 'col'.
// 'index' and 'stride' is a great design pattern that I will use always from now on, which i found in:
// https://devblogs.nvidia.com/even-easier-introduction-cuda/
// The amazing part is that you can change the block and dimension size without having to adjust the kernel,
// for performance comparison. It will automatically adjust and still compute all the necessary data.
#define RECONSTRUCTION_FUNCTION(name, code)                                                         \
template <typename scalar_t>                                                                        \
__global__ void name ## _cuda(                                                                      \
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> img,                    \
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t>  grad_x)          \
{                                                                                                   \
    int index = blockIdx.x * blockDim.x + threadIdx.x;                                              \
    int stride = blockDim.x * gridDim.x;                                                            \
    int col = blockIdx.y;                                                                           \
    code                                                                                            \
}


RECONSTRUCTION_FUNCTION(lr_kernel,{
    for(int y = index+1; y < img.size(1)-1; y+=stride){
        for(int x = 1; x < img.size(2)-1; x++){
            img[col][y][x] = (img[col][y][x] + img[col][y][x - 1] + grad_x[col][y][x - 1]) / 2;
        }
    }
})

RECONSTRUCTION_FUNCTION(rl_kernel,{
    for(int y = index+1; y < img.size(1)-1; y+=stride){
        for(int x = img.size(2)-2; x > 0; x--){
            img[col][y][x] = (img[col][y][x] + img[col][y][x + 1] - grad_x[col][y][x]) / 2;
        }
    }
})

RECONSTRUCTION_FUNCTION(tb_kernel,{
    for(int x = index+1; x < img.size(2)-1; x+=stride){
        for(int y = 1; y < img.size(1)-1; y++){
            img[col][y][x] = (img[col][y][x] + img[col][y-1][x] + grad_x[col][y-1][x]) / 2;
        }
    }
})

RECONSTRUCTION_FUNCTION(bt_kernel,{
    for(int x = index+1; x < img.size(2)-1; x+=stride){
        for(int y = img.size(1)-2; y > 0; y--){
            img[col][y][x] = (img[col][y][x] + img[col][y+1][x] - grad_x[col][y][x]) / 2;
        }
    }
})


void step_cuda(int step, torch::Tensor img, torch::Tensor grad){

    // Compute wavefront size. This is the number of parallel workers we have.
    // It is perpendicular to our walking direction, eg. the LR kernel has the height 'img' as wavefront.
    const int wavefront_size = (step%2==0)?img.size(1):img.size(2);

    // Compute number of blocks from wavefront
    const int blockSize = 1024;
    const int numBlocks = (wavefront_size + blockSize - 1) / blockSize;

    // Add a second dimension for the colors
    const dim3 numBlocksWithColors(numBlocks,img.size(0));

    // Automatically determine the data type of our CUDA kernels
    AT_DISPATCH_FLOATING_TYPES(img.type(), "step_cuda", ([&] {
        // Create accessors
        auto img_accessor = img.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>();
        auto grad_accessor = grad.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>();

        // The actual step
        switch(step%4){
            case 0:
                lr_kernel_cuda<scalar_t><<<numBlocksWithColors,blockSize>>>(img_accessor, grad_accessor);
                break;
            case 1:
                tb_kernel_cuda<scalar_t><<<numBlocksWithColors,blockSize>>>(img_accessor, grad_accessor);
                break;
            case 2:
                rl_kernel_cuda<scalar_t><<<numBlocksWithColors,blockSize>>>(img_accessor, grad_accessor);
                break;
            default:
                bt_kernel_cuda<scalar_t><<<numBlocksWithColors,blockSize>>>(img_accessor, grad_accessor);
                break;
        }
    }));
}
